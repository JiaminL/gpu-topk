#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <cub/util_allocator.cuh>

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>
#include <algorithm>
#include <numeric>
#include <unistd.h>

#include "printFunctions.cuh"
#include "generateProblems.cuh"
#include "sortTopK.cuh"
#include "radixSelectTopK.cuh"
#include "bitonicTopK.cuh"

#define IS_PRINT_EVERY_TESTING false
#define IS_PRINT_DIFF true

#define SETUP_TIMING()       \
    hipEvent_t start, stop; \
    hipEventCreate(&start); \
    hipEventCreate(&stop);

#define TIME_FUNC(f, t)                        \
    {                                          \
        hipEventRecord(start, 0);             \
        f;                                     \
        hipEventRecord(stop, 0);              \
        hipEventSynchronize(stop);            \
        hipEventElapsedTime(&t, start, stop); \
    }

#define NUMBEROFALGORITHMS 3
#define INIT_FUNCTIONS()                                                                           \
    typedef hipError_t (*ptrToTimingFunction)(KeyT*, uint, uint, KeyT*, CachingDeviceAllocator&); \
    const char* namesOfTimingFunctions[NUMBEROFALGORITHMS] = {                                     \
        "Sort",                                                                                    \
        "Radix Select",                                                                            \
        "Bitonic TopK",                                                                            \
    };                                                                                             \
    ptrToTimingFunction arrayOfTimingFunctions[NUMBEROFALGORITHMS] = {                             \
        &sortTopK<KeyT>,                                                                           \
        &radixSelectTopK<KeyT>,                                                                    \
        &bitonicTopK<KeyT>,                                                                \
    };

using namespace std;

CachingDeviceAllocator g_allocator(true);  // Caching allocator for device memory

template <typename KeyT>
void compareAlgorithms(uint size, uint k, uint numTests, uint* algorithmsToTest, uint generateType) {
    KeyT* d_vec;
    KeyT* d_vec_copy;
    KeyT* d_res;
    float timeArray[NUMBEROFALGORITHMS][numTests];
    double totalTimesPerAlgorithm[NUMBEROFALGORITHMS];
    float averageTimesPerAlgorithm[NUMBEROFALGORITHMS];
    float minTimesPerAlgorithm[NUMBEROFALGORITHMS];
    float maxTimesPerAlgorithm[NUMBEROFALGORITHMS];
    double standardPerAlgorithm[NUMBEROFALGORITHMS];  // standard deviation 标准差
    KeyT* resultsArray[NUMBEROFALGORITHMS][numTests];

    uint winnerArray[numTests];
    uint timesWon[NUMBEROFALGORITHMS];
    uint i, j, m, x;
    int runOrder[NUMBEROFALGORITHMS];

    unsigned long long seed;
    timeval t1;
    float runtime;

    for (i = 0; i < numTests; i++)
        for (j = 0; j < NUMBEROFALGORITHMS; j++)
            resultsArray[j][i] = new KeyT[k];

    SETUP_TIMING()

    typedef void (*ptrToGeneratingFunction)(KeyT*, uint, hiprandGenerator_t, CachingDeviceAllocator&);
    // these are the functions that can be called
    INIT_FUNCTIONS()

    ptrToGeneratingFunction* arrayOfGenerators;
    const char** namesOfGeneratingFunctions;
    // this is the array of names of functions that generate problems of this type, ie float, double, or uint
    namesOfGeneratingFunctions = returnNamesOfGenerators<KeyT>();
    arrayOfGenerators = (ptrToGeneratingFunction*)returnGenFunctions<KeyT>();

    // zero out the totals and times won
    bzero(totalTimesPerAlgorithm, NUMBEROFALGORITHMS * sizeof(uint));
    bzero(timesWon, NUMBEROFALGORITHMS * sizeof(uint));
    // allocate space for d_vec, and d_vec_copy
    hipMalloc(&d_vec, size * sizeof(KeyT));
    hipMalloc(&d_vec_copy, size * sizeof(KeyT));
    hipMalloc(&d_res, k * sizeof(KeyT));

    // create the random generator.
    hiprandGenerator_t generator;

    printf("The distribution is: %s\n", namesOfGeneratingFunctions[generateType]);
    for (i = 0; i < numTests; i++) {
        // hipDeviceReset();
        gettimeofday(&t1, NULL);
        seed = t1.tv_usec * t1.tv_sec;

        for (m = 0; m < NUMBEROFALGORITHMS; m++) {
            runOrder[m] = m;
        }
        std::random_shuffle(runOrder, runOrder + NUMBEROFALGORITHMS);

        hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(generator, seed);
        // hiprandSetPseudoRandomGeneratorSeed(generator, 0);

#if IS_PRINT_EVERY_TESTING
        printf("Running test %u of %u for size: %u and k: %u\n", i + 1, numTests, size, k);
#endif
        // generate the random vector using the specified distribution
        arrayOfGenerators[generateType](d_vec, size, generator, g_allocator);

        // KeyT* h_vec = new KeyT[size];
        // hipMemcpy(h_vec, d_vec, size * sizeof(KeyT), hipMemcpyDeviceToHost);
        // h_vec[0] = (KeyT)(-1034);
        // h_vec[1] = (KeyT)(0.5);
        // h_vec[2] = (KeyT)(1.0);
        // hipMemcpy(d_vec, h_vec, size * sizeof(KeyT), hipMemcpyHostToDevice);
        // delete[] h_vec;

        // copy the vector to d_vec_copy, which will be used to restore it later
        hipMemcpy(d_vec_copy, d_vec, size * sizeof(KeyT), hipMemcpyDeviceToDevice);

        winnerArray[i] = 0;
        float currentWinningTime = INFINITY;
        // run the various timing functions
        for (x = 0; x < NUMBEROFALGORITHMS; x++) {
            j = runOrder[x];
            if (algorithmsToTest[j]) {
                // run timing function j
                TIME_FUNC(arrayOfTimingFunctions[j](d_vec_copy, size, k, d_res, g_allocator), runtime);
                // 我猜测 GPU 释放空间与函数返回是异步的，上一次测试申请的空间还没有释放结束，下一次测试函数就开始了
                // 由于我的 GPU 显存只有 8GB，如果原始数据大小为 2GB，因为 GPU 没有更多的 2GB 空间用来分配（d_vec_copy, d_vec 已经使用了 4GB）
                // 下一次测试必须等待，导致除第一个上 GPU 的测试外，其余测试都有 30 ~ 50 ms 不等的延时
                if (size == (uint)(2 << 30) / sizeof(KeyT)) {         // 2GB
                    // printf("sleep 100\n");
                    usleep(100000);                             // sleep 100 ms
                } else if (size == (uint)(1 << 30) / sizeof(KeyT)) {  // 1GB
                    usleep(50000);                              // sleep 50 ms
                }
#if IS_PRINT_EVERY_TESTING
                printf("\tTESTING: %-2u %-20s runtime: %f ms\n", j, namesOfTimingFunctions[j], runtime);
#endif

                // check for error
                hipError_t error = hipGetLastError();
                if (error != hipSuccess) {
                    // print the CUDA error message and exit
                    printf("CUDA error: %s\n", hipGetErrorString(error));
                    exit(-1);
                }

                // record the time result
                timeArray[j][i] = runtime;

                // record the value returned
                hipMemcpy(resultsArray[j][i], d_res, k * sizeof(KeyT), hipMemcpyDeviceToHost);
                std::sort(resultsArray[j][i], resultsArray[j][i] + k, std::greater<KeyT>());

                // update the current "winner" if necessary
                if (timeArray[j][i] < currentWinningTime) {
                    currentWinningTime = runtime;
                    winnerArray[i] = j;
                }

                // perform clean up
                hipMemcpy(d_vec_copy, d_vec, size * sizeof(KeyT), hipMemcpyDeviceToDevice);
                hipMemset(d_res, 0, k * sizeof(KeyT));
            }
        }
        hiprandDestroyGenerator(generator);
    }

    // calculate the statistical data
    fill_n(standardPerAlgorithm, NUMBEROFALGORITHMS, 0);
    for (j = 0; j < NUMBEROFALGORITHMS; j++) {
        maxTimesPerAlgorithm[j] = *max_element(timeArray[j], timeArray[j] + numTests);
        minTimesPerAlgorithm[j] = *min_element(timeArray[j], timeArray[j] + numTests);
        totalTimesPerAlgorithm[j] = accumulate(timeArray[j], timeArray[j] + numTests, 0.0);
        // 计算均值
        averageTimesPerAlgorithm[j] = totalTimesPerAlgorithm[j] / numTests;
        // 计算方差
        if (numTests > 1) {
            for (i = 0; i < numTests; i++) {
                standardPerAlgorithm[j] += pow(timeArray[j][i] - averageTimesPerAlgorithm[j], 2);
            }
            standardPerAlgorithm[j] = sqrt(standardPerAlgorithm[j] / (numTests - 1));
        }
    }

    // count the number of times each algorithm won.
    for (i = 0; i < numTests; i++) {
        timesWon[winnerArray[i]]++;
    }

#if IS_PRINT_EVERY_TESTING
    printf("\n\n");
#endif

    // print out the statistical data
    int total_algorithms = accumulate(algorithmsToTest, algorithmsToTest + NUMBEROFALGORITHMS, 0);
    // print out header of the table
    printf("%-20s %-15s %-15s %-15s", "algorithm", "minimum (ms)", "maximum (ms)", "average (ms)");
    if (numTests > 1) printf(" %-15s", "std dev");
    if (total_algorithms > 1) printf(" %-15s", "won times");
    printf("\n");
    // print out data
    for (i = 0; i < NUMBEROFALGORITHMS; i++) {
        if (algorithmsToTest[i]) {
            printf("%-20s %-15f %-15f %-15f", namesOfTimingFunctions[i], minTimesPerAlgorithm[i],
                   maxTimesPerAlgorithm[i], averageTimesPerAlgorithm[i]);
            if (numTests > 1) printf(" %-15f", standardPerAlgorithm[i]);
            if (total_algorithms > 1) printf(" %-15d", timesWon[i]);
            printf("\n");
        }
    }
    printf("\n");
#if IS_PRINT_DIFF
    if (algorithmsToTest[0]) {
        for (i = 0; i < numTests; i++) {
            for (j = 1; j < NUMBEROFALGORITHMS; j++) {
                if (algorithmsToTest[j]) {
                    for (int m = 0; m < k; m++)
                        if (resultsArray[j][i][m] != resultsArray[0][i][m]) {
                            std::cout << namesOfTimingFunctions[j] << " did not return the correct answer on test" << i + 1 << std::endl;
                            std::cout << "Method:\t";
                            // PrintFunctions::printArray<KeyT>(resultsArray[j][i], k);
                            std::cout << "Sort:\t";
                            // PrintFunctions::printArray<KeyT>(resultsArray[0][i], k);
                            std::cout << "\n";
                            for (int l = 0; l < k; l++) {
                                std::cout << (KeyT)resultsArray[j][i][l] << "  " << (KeyT)resultsArray[0][i][l] << std::endl;
                            }
                            break;
                        }
                }
            }
        }
    }
#endif

    // free memory
    for (i = 0; i < numTests; i++)
        for (j = 0; j < NUMBEROFALGORITHMS; j++)
            delete[] resultsArray[j][i];
    hipFree(d_vec);
    hipFree(d_vec_copy);
    hipFree(d_res);
}

template <typename KeyT>
void runTests(uint generateType, int K, uint startPower, uint stopPower, uint timesToTestEachK = 3) {
    // Algorithms To Run
    // timeSort, timeRadixSelect, timeBitonicTopK
    uint algorithmsToRun[NUMBEROFALGORITHMS];
    fill_n(algorithmsToRun, NUMBEROFALGORITHMS, 1);
    uint size;
    uint power;
    for (size = (1 << startPower), power = startPower; power <= stopPower; size <<= 1, power++) {
        printf("NOW STARTING A NEW TOP-K [size: 2^%u (%u), k: %d]\n", power, size, K);
        compareAlgorithms<KeyT>(size, K, timesToTestEachK, algorithmsToRun, generateType);
    }
}

int main(int argc, char** argv) {
    uint testCount;
    int K;
    uint type, distributionType, startPower, stopPower;
    if (argc == 7) {
        type = atoi(argv[1]);
        distributionType = atoi(argv[2]);
        K = atoi(argv[3]);
        testCount = atoi(argv[4]);
        startPower = atoi(argv[5]);
        stopPower = atoi(argv[6]);
    } else {
        printf("Please enter the type of value you want to test:\n1-float\n2-double\n3-uint\n");
        cin >> type;
        printf("Please enter distribution type: ");
        cin >> distributionType;
        printf("Please enter K: ");
        cin >> K;
        printf("Please enter number of tests to run per K: ");
        cin >> testCount;
        printf("Please enter start power (dataset size starts at 2^start)(max val: 29): ");
        cin >> startPower;
        printf("Please enter stop power (dataset size stops at 2^stop)(max val: 29): ");
        cin >> stopPower;
    }

    switch (type) {
        case 1:
            runTests<float>(distributionType, K, startPower, stopPower, testCount);
            break;
        case 2:
            // runTests<double>(distributionType,K,startPower,stopPower,testCount);
            break;
        case 3:
            runTests<unsigned int>(distributionType, K, startPower, stopPower, testCount);
            break;
        default:
            printf("You entered and invalid option, now exiting\n");
            break;
    }

    return 0;
}
